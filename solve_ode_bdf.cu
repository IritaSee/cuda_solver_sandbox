#include <iostream>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <vector>

double *ALGEBRAIC;
double *CONSTANTS;
double *RATES;
double *STATES;

__global__ void initConsts(double* CONSTANTS, double* RATES, double *STATES)
{
STATES[0] = -86.2;
CONSTANTS[0] = 8.314;
CONSTANTS[1] = 310;
CONSTANTS[2] = 96.485;
CONSTANTS[3] = 185;
CONSTANTS[4] = 16404;
CONSTANTS[5] = 10;
CONSTANTS[6] = 1000;
CONSTANTS[7] = 1;
CONSTANTS[8] = -52;
CONSTANTS[9] = 0.03;
CONSTANTS[10] = 5.4;
CONSTANTS[11] = 140;
STATES[1] = 138.3;
STATES[2] = 11.6;
CONSTANTS[12] = 2;
STATES[3] = 0.0002;
CONSTANTS[13] = 5.405;
CONSTANTS[14] = 0.096;
STATES[4] = 0;
STATES[5] = 1;
CONSTANTS[15] = 0.062;
STATES[6] = 0;
CONSTANTS[16] = 14.838;
STATES[7] = 0;
STATES[8] = 0.75;
STATES[9] = 0.75;
CONSTANTS[17] = 0.00029;
CONSTANTS[18] = 0.175;
STATES[10] = 0;
STATES[11] = 1;
STATES[12] = 1;
CONSTANTS[19] = 0.000592;
CONSTANTS[20] = 0.294;
STATES[13] = 1;
STATES[14] = 0;
CONSTANTS[21] = 1.362;
CONSTANTS[22] = 1;
CONSTANTS[23] = 40;
CONSTANTS[24] = 1000;
CONSTANTS[25] = 0.1;
CONSTANTS[26] = 2.5;
CONSTANTS[27] = 0.35;
CONSTANTS[28] = 1.38;
CONSTANTS[29] = 87.5;
CONSTANTS[30] = 0.825;
CONSTANTS[31] = 0.0005;
CONSTANTS[32] = 0.0146;
STATES[15] = 0.2;
STATES[16] = 1;
CONSTANTS[33] = 2;
CONSTANTS[34] = 0.016464;
CONSTANTS[35] = 0.25;
CONSTANTS[36] = 0.008232;
CONSTANTS[37] = 0.00025;
CONSTANTS[38] = 8e-5;
CONSTANTS[39] = 0.000425;
CONSTANTS[40] = 0.15;
CONSTANTS[41] = 0.001;
CONSTANTS[42] = 10;
CONSTANTS[43] = 0.3;
CONSTANTS[44] = 1094;
CONSTANTS[45] = 2.00000;
}

__device__ void computeRates(double VOI, double* CONSTANTS, double* RATES, double* STATES, double* ALGEBRAIC)
{
ALGEBRAIC[8] = 1.00000/(1.00000+exp((STATES[0]+20.0000)/7.00000));
ALGEBRAIC[21] =  1125.00*exp(- pow(STATES[0]+27.0000, 2.00000)/240.000)+80.0000+165.000/(1.00000+exp((25.0000 - STATES[0])/10.0000));
RATES[11] = (ALGEBRAIC[8] - STATES[11])/ALGEBRAIC[21];
ALGEBRAIC[10] = 1.00000/(1.00000+exp((STATES[0]+20.0000)/5.00000));
ALGEBRAIC[23] =  85.0000*exp(- pow(STATES[0]+45.0000, 2.00000)/320.000)+5.00000/(1.00000+exp((STATES[0] - 20.0000)/5.00000))+3.00000;
RATES[13] = (ALGEBRAIC[10] - STATES[13])/ALGEBRAIC[23];
ALGEBRAIC[11] = 1.00000/(1.00000+exp((20.0000 - STATES[0])/6.00000));
ALGEBRAIC[24] =  9.50000*exp(- pow(STATES[0]+40.0000, 2.00000)/1800.00)+0.800000;
RATES[14] = (ALGEBRAIC[11] - STATES[14])/ALGEBRAIC[24];
ALGEBRAIC[12] = (STATES[3]<0.000350000 ? 1.00000/(1.00000+pow(STATES[3]/0.000350000, 6.00000)) : 1.00000/(1.00000+pow(STATES[3]/0.000350000, 16.0000)));
ALGEBRAIC[25] = (ALGEBRAIC[12] - STATES[16])/CONSTANTS[33];
RATES[16] = (ALGEBRAIC[12]>STATES[16]&&STATES[0]>- 60.0000 ? 0.00000 : ALGEBRAIC[25]);
ALGEBRAIC[1] = 1.00000/(1.00000+exp((- 26.0000 - STATES[0])/7.00000));
ALGEBRAIC[14] = 450.000/(1.00000+exp((- 45.0000 - STATES[0])/10.0000));
ALGEBRAIC[27] = 6.00000/(1.00000+exp((STATES[0]+30.0000)/11.5000));
ALGEBRAIC[36] =  1.00000*ALGEBRAIC[14]*ALGEBRAIC[27];
RATES[4] = (ALGEBRAIC[1] - STATES[4])/ALGEBRAIC[36];
ALGEBRAIC[2] = 1.00000/(1.00000+exp((STATES[0]+88.0000)/24.0000));
ALGEBRAIC[15] = 3.00000/(1.00000+exp((- 60.0000 - STATES[0])/20.0000));
ALGEBRAIC[28] = 1.12000/(1.00000+exp((STATES[0] - 60.0000)/20.0000));
ALGEBRAIC[37] =  1.00000*ALGEBRAIC[15]*ALGEBRAIC[28];
RATES[5] = (ALGEBRAIC[2] - STATES[5])/ALGEBRAIC[37];
ALGEBRAIC[3] = 1.00000/(1.00000+exp((- 5.00000 - STATES[0])/14.0000));
ALGEBRAIC[16] = 1100.00/ pow((1.00000+exp((- 10.0000 - STATES[0])/6.00000)), 1.0 / 2);
ALGEBRAIC[29] = 1.00000/(1.00000+exp((STATES[0] - 60.0000)/20.0000));
ALGEBRAIC[38] =  1.00000*ALGEBRAIC[16]*ALGEBRAIC[29];
RATES[6] = (ALGEBRAIC[3] - STATES[6])/ALGEBRAIC[38];
ALGEBRAIC[4] = 1.00000/pow(1.00000+exp((- 56.8600 - STATES[0])/9.03000), 2.00000);
ALGEBRAIC[17] = 1.00000/(1.00000+exp((- 60.0000 - STATES[0])/5.00000));
ALGEBRAIC[30] = 0.100000/(1.00000+exp((STATES[0]+35.0000)/5.00000))+0.100000/(1.00000+exp((STATES[0] - 50.0000)/200.000));
ALGEBRAIC[39] =  1.00000*ALGEBRAIC[17]*ALGEBRAIC[30];
RATES[7] = (ALGEBRAIC[4] - STATES[7])/ALGEBRAIC[39];
ALGEBRAIC[5] = 1.00000/pow(1.00000+exp((STATES[0]+71.5500)/7.43000), 2.00000);
ALGEBRAIC[18] = (STATES[0]<- 40.0000 ?  0.0570000*exp(- (STATES[0]+80.0000)/6.80000) : 0.00000);
ALGEBRAIC[31] = (STATES[0]<- 40.0000 ?  2.70000*exp( 0.0790000*STATES[0])+ 310000.*exp( 0.348500*STATES[0]) : 0.770000/( 0.130000*(1.00000+exp((STATES[0]+10.6600)/- 11.1000))));
ALGEBRAIC[40] = 1.00000/(ALGEBRAIC[18]+ALGEBRAIC[31]);
RATES[8] = (ALGEBRAIC[5] - STATES[8])/ALGEBRAIC[40];
ALGEBRAIC[6] = 1.00000/pow(1.00000+exp((STATES[0]+71.5500)/7.43000), 2.00000);
ALGEBRAIC[19] = (STATES[0]<- 40.0000 ? (( ( - 25428.0*exp( 0.244400*STATES[0]) -  6.94800e-06*exp( - 0.0439100*STATES[0]))*(STATES[0]+37.7800))/1.00000)/(1.00000+exp( 0.311000*(STATES[0]+79.2300))) : 0.00000);
ALGEBRAIC[32] = (STATES[0]<- 40.0000 ? ( 0.0242400*exp( - 0.0105200*STATES[0]))/(1.00000+exp( - 0.137800*(STATES[0]+40.1400))) : ( 0.600000*exp( 0.0570000*STATES[0]))/(1.00000+exp( - 0.100000*(STATES[0]+32.0000))));
ALGEBRAIC[41] = 1.00000/(ALGEBRAIC[19]+ALGEBRAIC[32]);
RATES[9] = (ALGEBRAIC[6] - STATES[9])/ALGEBRAIC[41];
ALGEBRAIC[7] = 1.00000/(1.00000+exp((- 5.00000 - STATES[0])/7.50000));
ALGEBRAIC[20] = 1.40000/(1.00000+exp((- 35.0000 - STATES[0])/13.0000))+0.250000;
ALGEBRAIC[33] = 1.40000/(1.00000+exp((STATES[0]+5.00000)/5.00000));
ALGEBRAIC[42] = 1.00000/(1.00000+exp((50.0000 - STATES[0])/20.0000));
ALGEBRAIC[45] =  1.00000*ALGEBRAIC[20]*ALGEBRAIC[33]+ALGEBRAIC[42];
RATES[10] = (ALGEBRAIC[7] - STATES[10])/ALGEBRAIC[45];
ALGEBRAIC[9] = 1.00000/(1.00000+pow(STATES[3]/0.000325000, 8.00000));
ALGEBRAIC[22] = 0.100000/(1.00000+exp((STATES[3] - 0.000500000)/0.000100000));
ALGEBRAIC[34] = 0.200000/(1.00000+exp((STATES[3] - 0.000750000)/0.000800000));
ALGEBRAIC[43] = (ALGEBRAIC[9]+ALGEBRAIC[22]+ALGEBRAIC[34]+0.230000)/1.46000;
ALGEBRAIC[46] = (ALGEBRAIC[43] - STATES[12])/CONSTANTS[45];
RATES[12] = (ALGEBRAIC[43]>STATES[12]&&STATES[0]>- 60.0000 ? 0.00000 : ALGEBRAIC[46]);
ALGEBRAIC[58] = (( (( CONSTANTS[21]*CONSTANTS[10])/(CONSTANTS[10]+CONSTANTS[22]))*STATES[2])/(STATES[2]+CONSTANTS[23]))/(1.00000+ 0.124500*exp(( - 0.100000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))+ 0.0353000*exp(( - STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])));
ALGEBRAIC[13] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[11]/STATES[2]);
ALGEBRAIC[53] =  CONSTANTS[16]*pow(STATES[7], 3.00000)*STATES[8]*STATES[9]*(STATES[0] - ALGEBRAIC[13]);
ALGEBRAIC[54] =  CONSTANTS[17]*(STATES[0] - ALGEBRAIC[13]);
ALGEBRAIC[59] = ( CONSTANTS[24]*( exp(( CONSTANTS[27]*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))*pow(STATES[2], 3.00000)*CONSTANTS[12] -  exp(( (CONSTANTS[27] - 1.00000)*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))*pow(CONSTANTS[11], 3.00000)*STATES[3]*CONSTANTS[26]))/( (pow(CONSTANTS[29], 3.00000)+pow(CONSTANTS[11], 3.00000))*(CONSTANTS[28]+CONSTANTS[12])*(1.00000+ CONSTANTS[25]*exp(( (CONSTANTS[27] - 1.00000)*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))));
RATES[2] = ( - (ALGEBRAIC[53]+ALGEBRAIC[54]+ 3.00000*ALGEBRAIC[58]+ 3.00000*ALGEBRAIC[59])*CONSTANTS[3])/( CONSTANTS[4]*CONSTANTS[2]);
ALGEBRAIC[26] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[10]/STATES[1]);
ALGEBRAIC[47] = 0.100000/(1.00000+exp( 0.0600000*((STATES[0] - ALGEBRAIC[26]) - 200.000)));
ALGEBRAIC[48] = ( 3.00000*exp( 0.000200000*((STATES[0] - ALGEBRAIC[26])+100.000))+ 1.00000*exp( 0.100000*((STATES[0] - ALGEBRAIC[26]) - 10.0000)))/(1.00000+exp( - 0.500000*(STATES[0] - ALGEBRAIC[26])));
ALGEBRAIC[49] = ALGEBRAIC[47]/(ALGEBRAIC[47]+ALGEBRAIC[48]);
ALGEBRAIC[50] =  CONSTANTS[13]*ALGEBRAIC[49]* pow((CONSTANTS[10]/5.40000), 1.0 / 2)*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[57] =  CONSTANTS[20]*STATES[14]*STATES[13]*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[51] =  CONSTANTS[14]* pow((CONSTANTS[10]/5.40000), 1.0 / 2)*STATES[4]*STATES[5]*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[35] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log((CONSTANTS[10]+ CONSTANTS[9]*CONSTANTS[11])/(STATES[1]+ CONSTANTS[9]*STATES[2]));
ALGEBRAIC[52] =  CONSTANTS[15]*pow(STATES[6], 2.00000)*(STATES[0] - ALGEBRAIC[35]);
ALGEBRAIC[55] = ( (( CONSTANTS[18]*STATES[10]*STATES[11]*STATES[12]*4.00000*STATES[0]*pow(CONSTANTS[2], 2.00000))/( CONSTANTS[0]*CONSTANTS[1]))*( STATES[3]*exp(( 2.00000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])) -  0.341000*CONSTANTS[12]))/(exp(( 2.00000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])) - 1.00000);
ALGEBRAIC[44] =  (( 0.500000*CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[12]/STATES[3]);
ALGEBRAIC[56] =  CONSTANTS[19]*(STATES[0] - ALGEBRAIC[44]);
ALGEBRAIC[61] = ( CONSTANTS[32]*(STATES[0] - ALGEBRAIC[26]))/(1.00000+exp((25.0000 - STATES[0])/5.98000));
ALGEBRAIC[60] = ( CONSTANTS[30]*STATES[3])/(STATES[3]+CONSTANTS[31]);
ALGEBRAIC[0] = (VOI -  floor(VOI/CONSTANTS[6])*CONSTANTS[6]>=CONSTANTS[5]&&VOI -  floor(VOI/CONSTANTS[6])*CONSTANTS[6]<=CONSTANTS[5]+CONSTANTS[7] ? CONSTANTS[8] : 0.00000);
RATES[0] = - (ALGEBRAIC[50]+ALGEBRAIC[57]+ALGEBRAIC[51]+ALGEBRAIC[52]+ALGEBRAIC[55]+ALGEBRAIC[58]+ALGEBRAIC[53]+ALGEBRAIC[54]+ALGEBRAIC[59]+ALGEBRAIC[56]+ALGEBRAIC[61]+ALGEBRAIC[60]+ALGEBRAIC[0]);
RATES[1] = ( - ((ALGEBRAIC[50]+ALGEBRAIC[57]+ALGEBRAIC[51]+ALGEBRAIC[52]+ALGEBRAIC[61]+ALGEBRAIC[0]) -  2.00000*ALGEBRAIC[58])*CONSTANTS[3])/( CONSTANTS[4]*CONSTANTS[2]);
ALGEBRAIC[62] =  (( CONSTANTS[34]*pow(STATES[15], 2.00000))/(pow(CONSTANTS[35], 2.00000)+pow(STATES[15], 2.00000))+CONSTANTS[36])*STATES[10]*STATES[16];
ALGEBRAIC[63] = CONSTANTS[39]/(1.00000+pow(CONSTANTS[37], 2.00000)/pow(STATES[3], 2.00000));
ALGEBRAIC[64] =  CONSTANTS[38]*(STATES[15] - STATES[3]);
ALGEBRAIC[65] = (( (- ((ALGEBRAIC[55]+ALGEBRAIC[56]+ALGEBRAIC[60]) -  2.00000*ALGEBRAIC[59])/( 2.00000*CONSTANTS[4]*CONSTANTS[2]))*CONSTANTS[3]+ALGEBRAIC[64]) - ALGEBRAIC[63])+ALGEBRAIC[62];
ALGEBRAIC[67] = 1.00000/(1.00000+( CONSTANTS[40]*CONSTANTS[41])/pow(STATES[3]+CONSTANTS[41], 2.00000));
RATES[3] =  ALGEBRAIC[65]*ALGEBRAIC[67];
ALGEBRAIC[66] =  (CONSTANTS[4]/CONSTANTS[44])*(ALGEBRAIC[63] - (ALGEBRAIC[62]+ALGEBRAIC[64]));
ALGEBRAIC[68] = 1.00000/(1.00000+( CONSTANTS[42]*CONSTANTS[43])/pow(STATES[15]+CONSTANTS[43], 2.00000));
RATES[15] =  ALGEBRAIC[66]*ALGEBRAIC[68];
}

__device__ void computeVariables(double VOI, double* CONSTANTS, double* RATES, double* STATES, double* ALGEBRAIC)
{
ALGEBRAIC[8] = 1.00000/(1.00000+exp((STATES[0]+20.0000)/7.00000));
ALGEBRAIC[21] =  1125.00*exp(- pow(STATES[0]+27.0000, 2.00000)/240.000)+80.0000+165.000/(1.00000+exp((25.0000 - STATES[0])/10.0000));
ALGEBRAIC[10] = 1.00000/(1.00000+exp((STATES[0]+20.0000)/5.00000));
ALGEBRAIC[23] =  85.0000*exp(- pow(STATES[0]+45.0000, 2.00000)/320.000)+5.00000/(1.00000+exp((STATES[0] - 20.0000)/5.00000))+3.00000;
ALGEBRAIC[11] = 1.00000/(1.00000+exp((20.0000 - STATES[0])/6.00000));
ALGEBRAIC[24] =  9.50000*exp(- pow(STATES[0]+40.0000, 2.00000)/1800.00)+0.800000;
ALGEBRAIC[12] = (STATES[3]<0.000350000 ? 1.00000/(1.00000+pow(STATES[3]/0.000350000, 6.00000)) : 1.00000/(1.00000+pow(STATES[3]/0.000350000, 16.0000)));
ALGEBRAIC[25] = (ALGEBRAIC[12] - STATES[16])/CONSTANTS[33];
ALGEBRAIC[1] = 1.00000/(1.00000+exp((- 26.0000 - STATES[0])/7.00000));
ALGEBRAIC[14] = 450.000/(1.00000+exp((- 45.0000 - STATES[0])/10.0000));
ALGEBRAIC[27] = 6.00000/(1.00000+exp((STATES[0]+30.0000)/11.5000));
ALGEBRAIC[36] =  1.00000*ALGEBRAIC[14]*ALGEBRAIC[27];
ALGEBRAIC[2] = 1.00000/(1.00000+exp((STATES[0]+88.0000)/24.0000));
ALGEBRAIC[15] = 3.00000/(1.00000+exp((- 60.0000 - STATES[0])/20.0000));
ALGEBRAIC[28] = 1.12000/(1.00000+exp((STATES[0] - 60.0000)/20.0000));
ALGEBRAIC[37] =  1.00000*ALGEBRAIC[15]*ALGEBRAIC[28];
ALGEBRAIC[3] = 1.00000/(1.00000+exp((- 5.00000 - STATES[0])/14.0000));
ALGEBRAIC[16] = 1100.00/ pow((1.00000+exp((- 10.0000 - STATES[0])/6.00000)), 1.0 / 2);
ALGEBRAIC[29] = 1.00000/(1.00000+exp((STATES[0] - 60.0000)/20.0000));
ALGEBRAIC[38] =  1.00000*ALGEBRAIC[16]*ALGEBRAIC[29];
ALGEBRAIC[4] = 1.00000/pow(1.00000+exp((- 56.8600 - STATES[0])/9.03000), 2.00000);
ALGEBRAIC[17] = 1.00000/(1.00000+exp((- 60.0000 - STATES[0])/5.00000));
ALGEBRAIC[30] = 0.100000/(1.00000+exp((STATES[0]+35.0000)/5.00000))+0.100000/(1.00000+exp((STATES[0] - 50.0000)/200.000));
ALGEBRAIC[39] =  1.00000*ALGEBRAIC[17]*ALGEBRAIC[30];
ALGEBRAIC[5] = 1.00000/pow(1.00000+exp((STATES[0]+71.5500)/7.43000), 2.00000);
ALGEBRAIC[18] = (STATES[0]<- 40.0000 ?  0.0570000*exp(- (STATES[0]+80.0000)/6.80000) : 0.00000);
ALGEBRAIC[31] = (STATES[0]<- 40.0000 ?  2.70000*exp( 0.0790000*STATES[0])+ 310000.*exp( 0.348500*STATES[0]) : 0.770000/( 0.130000*(1.00000+exp((STATES[0]+10.6600)/- 11.1000))));
ALGEBRAIC[40] = 1.00000/(ALGEBRAIC[18]+ALGEBRAIC[31]);
ALGEBRAIC[6] = 1.00000/pow(1.00000+exp((STATES[0]+71.5500)/7.43000), 2.00000);
ALGEBRAIC[19] = (STATES[0]<- 40.0000 ? (( ( - 25428.0*exp( 0.244400*STATES[0]) -  6.94800e-06*exp( - 0.0439100*STATES[0]))*(STATES[0]+37.7800))/1.00000)/(1.00000+exp( 0.311000*(STATES[0]+79.2300))) : 0.00000);
ALGEBRAIC[32] = (STATES[0]<- 40.0000 ? ( 0.0242400*exp( - 0.0105200*STATES[0]))/(1.00000+exp( - 0.137800*(STATES[0]+40.1400))) : ( 0.600000*exp( 0.0570000*STATES[0]))/(1.00000+exp( - 0.100000*(STATES[0]+32.0000))));
ALGEBRAIC[41] = 1.00000/(ALGEBRAIC[19]+ALGEBRAIC[32]);
ALGEBRAIC[7] = 1.00000/(1.00000+exp((- 5.00000 - STATES[0])/7.50000));
ALGEBRAIC[20] = 1.40000/(1.00000+exp((- 35.0000 - STATES[0])/13.0000))+0.250000;
ALGEBRAIC[33] = 1.40000/(1.00000+exp((STATES[0]+5.00000)/5.00000));
ALGEBRAIC[42] = 1.00000/(1.00000+exp((50.0000 - STATES[0])/20.0000));
ALGEBRAIC[45] =  1.00000*ALGEBRAIC[20]*ALGEBRAIC[33]+ALGEBRAIC[42];
ALGEBRAIC[9] = 1.00000/(1.00000+pow(STATES[3]/0.000325000, 8.00000));
ALGEBRAIC[22] = 0.100000/(1.00000+exp((STATES[3] - 0.000500000)/0.000100000));
ALGEBRAIC[34] = 0.200000/(1.00000+exp((STATES[3] - 0.000750000)/0.000800000));
ALGEBRAIC[43] = (ALGEBRAIC[9]+ALGEBRAIC[22]+ALGEBRAIC[34]+0.230000)/1.46000;
ALGEBRAIC[46] = (ALGEBRAIC[43] - STATES[12])/CONSTANTS[45];
ALGEBRAIC[58] = (( (( CONSTANTS[21]*CONSTANTS[10])/(CONSTANTS[10]+CONSTANTS[22]))*STATES[2])/(STATES[2]+CONSTANTS[23]))/(1.00000+ 0.124500*exp(( - 0.100000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))+ 0.0353000*exp(( - STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])));
ALGEBRAIC[13] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[11]/STATES[2]);
ALGEBRAIC[53] =  CONSTANTS[16]*pow(STATES[7], 3.00000)*STATES[8]*STATES[9]*(STATES[0] - ALGEBRAIC[13]);
ALGEBRAIC[54] =  CONSTANTS[17]*(STATES[0] - ALGEBRAIC[13]);
ALGEBRAIC[59] = ( CONSTANTS[24]*( exp(( CONSTANTS[27]*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))*pow(STATES[2], 3.00000)*CONSTANTS[12] -  exp(( (CONSTANTS[27] - 1.00000)*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))*pow(CONSTANTS[11], 3.00000)*STATES[3]*CONSTANTS[26]))/( (pow(CONSTANTS[29], 3.00000)+pow(CONSTANTS[11], 3.00000))*(CONSTANTS[28]+CONSTANTS[12])*(1.00000+ CONSTANTS[25]*exp(( (CONSTANTS[27] - 1.00000)*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))));
ALGEBRAIC[26] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[10]/STATES[1]);
ALGEBRAIC[47] = 0.100000/(1.00000+exp( 0.0600000*((STATES[0] - ALGEBRAIC[26]) - 200.000)));
ALGEBRAIC[48] = ( 3.00000*exp( 0.000200000*((STATES[0] - ALGEBRAIC[26])+100.000))+ 1.00000*exp( 0.100000*((STATES[0] - ALGEBRAIC[26]) - 10.0000)))/(1.00000+exp( - 0.500000*(STATES[0] - ALGEBRAIC[26])));
ALGEBRAIC[49] = ALGEBRAIC[47]/(ALGEBRAIC[47]+ALGEBRAIC[48]);
ALGEBRAIC[50] =  CONSTANTS[13]*ALGEBRAIC[49]* pow((CONSTANTS[10]/5.40000), 1.0 / 2)*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[57] =  CONSTANTS[20]*STATES[14]*STATES[13]*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[51] =  CONSTANTS[14]* pow((CONSTANTS[10]/5.40000), 1.0 / 2)*STATES[4]*STATES[5]*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[35] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log((CONSTANTS[10]+ CONSTANTS[9]*CONSTANTS[11])/(STATES[1]+ CONSTANTS[9]*STATES[2]));
ALGEBRAIC[52] =  CONSTANTS[15]*pow(STATES[6], 2.00000)*(STATES[0] - ALGEBRAIC[35]);
ALGEBRAIC[55] = ( (( CONSTANTS[18]*STATES[10]*STATES[11]*STATES[12]*4.00000*STATES[0]*pow(CONSTANTS[2], 2.00000))/( CONSTANTS[0]*CONSTANTS[1]))*( STATES[3]*exp(( 2.00000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])) -  0.341000*CONSTANTS[12]))/(exp(( 2.00000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])) - 1.00000);
ALGEBRAIC[44] =  (( 0.500000*CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[12]/STATES[3]);
ALGEBRAIC[56] =  CONSTANTS[19]*(STATES[0] - ALGEBRAIC[44]);
ALGEBRAIC[61] = ( CONSTANTS[32]*(STATES[0] - ALGEBRAIC[26]))/(1.00000+exp((25.0000 - STATES[0])/5.98000));
ALGEBRAIC[60] = ( CONSTANTS[30]*STATES[3])/(STATES[3]+CONSTANTS[31]);
ALGEBRAIC[0] = (VOI -  floor(VOI/CONSTANTS[6])*CONSTANTS[6]>=CONSTANTS[5]&&VOI -  floor(VOI/CONSTANTS[6])*CONSTANTS[6]<=CONSTANTS[5]+CONSTANTS[7] ? CONSTANTS[8] : 0.00000);
ALGEBRAIC[62] =  (( CONSTANTS[34]*pow(STATES[15], 2.00000))/(pow(CONSTANTS[35], 2.00000)+pow(STATES[15], 2.00000))+CONSTANTS[36])*STATES[10]*STATES[16];
ALGEBRAIC[63] = CONSTANTS[39]/(1.00000+pow(CONSTANTS[37], 2.00000)/pow(STATES[3], 2.00000));
ALGEBRAIC[64] =  CONSTANTS[38]*(STATES[15] - STATES[3]);
ALGEBRAIC[65] = (( (- ((ALGEBRAIC[55]+ALGEBRAIC[56]+ALGEBRAIC[60]) -  2.00000*ALGEBRAIC[59])/( 2.00000*CONSTANTS[4]*CONSTANTS[2]))*CONSTANTS[3]+ALGEBRAIC[64]) - ALGEBRAIC[63])+ALGEBRAIC[62];
ALGEBRAIC[67] = 1.00000/(1.00000+( CONSTANTS[40]*CONSTANTS[41])/pow(STATES[3]+CONSTANTS[41], 2.00000));
ALGEBRAIC[66] =  (CONSTANTS[4]/CONSTANTS[44])*(ALGEBRAIC[63] - (ALGEBRAIC[62]+ALGEBRAIC[64]));
ALGEBRAIC[68] = 1.00000/(1.00000+( CONSTANTS[42]*CONSTANTS[43])/pow(STATES[15]+CONSTANTS[43], 2.00000));
}

// Define the function f(t, y) for the system of ODEs dy/dt = f(t, y)
// __global__ void ODEFunction(double *y, double *dy, double t, int n) {
//     // ODE system: dy/dt = A * y, where A is a diagonal matrix with different coefficients
//     for (int i = 0; i < n; ++i) {
//         switch (i) {
//             case 0: dy[i] = -2.0 * y[i]; break;   // dy1/dt = -2 * y1
//             case 1: dy[i] = -1.0 * y[i]; break;   // dy2/dt = -y2
//             case 2: dy[i] = -0.5 * y[i]; break;   // dy3/dt = -0.5 * y3
//             case 3: dy[i] = -0.25 * y[i]; break;  // dy4/dt = -0.25 * y4
//             case 4: dy[i] = -0.1 * y[i]; break;   // dy5/dt = -0.1 * y5
//         }
//     }
// }

__global__ void bridgeFunction(double t, double* STATES, double* RATES, double* CONSTANTS, double* ALGEBRAIC){
    
    computeRates(t, CONSTANTS, RATES, STATES, ALGEBRAIC);

    computeVariables(t, CONSTANTS, RATES, STATES, ALGEBRAIC);

}


// Solve the system of ODEs using BDF
void solveODEBDF(double t0, double t1, double* STATES, double* RATES, double* CONSTANTS, double* ALGEBRAIC, int n, int steps, bool init) {
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    int num_of_constants = 45;
    int num_of_states = 17;
    int num_of_algebraic = 69;
    int num_of_rates = 17;

    double dt = (t1 - t0) / steps;
    double t = t0;

    // Allocate memory for y and dy
    double *d_ALGEBRAIC;
    double *d_CONSTANTS;
    double *d_RATES;
    double *d_STATES;

    hipMalloc(&d_ALGEBRAIC, num_of_algebraic * sizeof(double));
    hipMalloc(&d_CONSTANTS, num_of_constants * sizeof(double));
    hipMalloc(&d_RATES, num_of_rates * sizeof(double));
    hipMalloc(&d_STATES, num_of_states * sizeof(double));

    // Initialize y with initial conditions
    // cudaMemcpy(d_y, y0, n * sizeof(double), cudaMemcpyHostToDevice);
    if (init == true){
    initConsts<<<1,n>>>(CONSTANTS, RATES, STATES);
    hipDeviceSynchronize();
    // init == true;
    }
    
    for (int i = 0; i < steps; ++i) {
        // Compute dy = f(t, y)
        // ODEFunction<<<1, n>>>(d_y, d_dy, t, n);
        bridgeFunction<<<1,n>>>(t, STATES, RATES, CONSTANTS, ALGEBRAIC);
        hipDeviceSynchronize();

        // Solve the linear system (I - dt * J) * y_new = y_old
        // For this example, we assume J is identity and dy is small, so BDF simplifies to:
        // y_new = y_old + dt * f(t, y)

        double alpha = dt;
        hipblasDaxpy(cublasHandle, n, &alpha, RATES, 1, STATES, 1); // y_new = y_old + dt * dy

        // Update time
        t += dt;
    }

    // Copy the result back to host
    hipMemcpy(STATES, d_STATES, n * sizeof(double), hipMemcpyDeviceToHost);
    

    // Clean up
    hipFree(d_ALGEBRAIC);
    hipFree(d_CONSTANTS);
    hipFree(d_STATES);
    hipFree(d_RATES);

    hipblasDestroy(cublasHandle);
}



int main() {
    double t0 = 0.0;
    double t1 = 1.0;
    int n = 17;  // Number of ODEs
    int steps = 10;
    bool init = true;

    int num_of_constants = 45;
    int num_of_states = 17;
    int num_of_algebraic = 69;
    int num_of_rates = 17;

    STATES = (double *)malloc(num_of_states  * sizeof(double));
    RATES = (double *)malloc(num_of_rates  * sizeof(double));
    ALGEBRAIC = (double *)malloc(num_of_algebraic  * sizeof(double));
    CONSTANTS = (double *)malloc(num_of_constants  * sizeof(double));

    // Initial conditions for the 5 ODEs
    // std::vector<double> y0 = {1.0, 1.0, 1.0, 1.0, 1.0};
    for(int loop=0; loop < 100; loop++){
        solveODEBDF(t0, t1, STATES, RATES, CONSTANTS, ALGEBRAIC, n, steps, init);
        init = false;
        for (int i = 0; i < n; ++i) {
        std::cout << "rates " << i << " = " << RATES[i] << std::endl;
    }
        printf("\n");
        t0 = t1;
        t1 = t1+0.5;
    }
    

    // Output the solutions at t = t1
    // std::cout << "Solutions at t = " << t1 << " are:" << std::endl;
    // for (int i = 0; i < n; ++i) {
    //     std::cout << "y" << i + 1 << " = " << y0[i] << std::endl;
    // }

    return 0;
}
